#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#define N 256 //Default matrix size NxN
#define A(i,j) A[(i)*cols+(j)]  // row-major layout
#define C(i,j) C[(i)*cols+(j)]  // row-major layout

__global__ void convolution(int *A, int *C)
{
	//Filter
	int filter[3][3] = { { 1, 2, 1 },{ 2, 4, 2 },{ 1, 2, 1 } };

	//Needs for row-major layout
	int cols = N + 2;
	//int i = blockIdx.y * blockDim.y + threadIdx.y;
	for (int row = blockIdx.y * blockDim.y + threadIdx.y; row < N + 2; row += blockDim.x * gridDim.x) {
		for (int col = blockIdx.x * blockDim.x + threadIdx.x; col < N + 2; col += blockDim.y * gridDim.y) {
			int value = 99999;

			if (0 < row && row < N + 1 && 0 < col && col < N + 1)
			{
				value = 0;
				value = value + A(row - 1, col - 1)	*  filter[0][0];
				value = value + A(row - 1, col)		*  filter[0][1];
				value = value + A(row - 1, col + 1)	*  filter[0][2];
				value = value + A(row, col - 1)		*  filter[1][0];
				value = value + A(row, col)			*  filter[1][1];
				value = value + A(row, col + 1)		*  filter[1][2];
				value = value + A(row + 1, col - 1)	*  filter[2][0];
				value = value + A(row + 1, col)		*  filter[2][1];
				value = value + A(row + 1, col + 1)	*  filter[2][2];
			}
			C(row, col) = value;
		}
	}

}

#define BLOCK_SIZE 16

int main(void)
{
	//Host variables
	int A[N + 2][N + 2] = {};//+2 for padding matrix
	int *C;

	//Device variables
	int *A_d = 0, *C_d = 0;

	//Needs for row-major layout
	int cols = N + 2;

	//Calculate memory size 
	int memorySize = (N + 2) * (N + 2);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Init matrix by 0
	for (int i = 0; i < N + 2; i++) {
		for (int j = 0; j < N + 2; j++) {
			A[i][j] = 0;
		}
	}

	//Generate random values between 0 and 9
	srand(time(NULL));
	for (int i = 1; i < N + 2; i++) {
		for (int j = 1; j < N + 2; j++) {
			A[i][j] = rand() % 10;
		}
	}

	C = (int *)malloc(sizeof(*C)*memorySize);

	hipMalloc((void**)&A_d, sizeof(*A_d)*memorySize);
	hipMalloc((void**)&C_d, sizeof(*C_d)*memorySize);

	//Copy from host to device
	hipMemcpy(A_d, A, sizeof(*A_d)*memorySize, hipMemcpyHostToDevice);
	// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((N + 2) / dimBlock.x, (N + 2) / dimBlock.y);
	printf("%d, %d \n", dimGrid.x, dimGrid.y);
	hipEventRecord(start);
	convolution << <dimGrid, dimBlock >> > (A_d, C_d);//Block-thread
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	//Copy from device to host
	hipMemcpy(C, C_d, sizeof(*C)*memorySize, hipMemcpyDeviceToHost);

	////Print result
	for (int i = 0; i < N + 2; i++) {
		for (int j = 0; j < N + 2; j++) {
			printf("%d ", C(i, j));
		}
		printf("\n");
	}

	//Free memory
	hipFree(C_d);
	hipFree(A_d);
	free(C);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f", milliseconds);
	return EXIT_SUCCESS;
}